#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "customizable_conv.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

void forward_custom_conv_layer_gpu(layer l, network net)
{
    fill_gpu(l.outputs*l.batch, 0, l.output_gpu, 1);

    // float one = 1;

    im2col_gpu(net.input_gpu, l.c, l.h, l.w, l.size, l.stride, l.pad, l.output_gpu);

    add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    activate_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation);
}

void pull_custom_conv_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
}

void push_custom_conv_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.nweights);
    cuda_push_array(l.biases_gpu, l.biases, l.n);
}