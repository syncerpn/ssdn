#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include "hiprand/hiprand_kernel.h"

#include "blas.h"
#include "hip/hip_runtime.h"
#include "utils.h"

#include <assert.h>

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) X[i*INCX] = ALPHA;
}

void fill_gpu(int N, float ALPHA, float *X, int INCX) {
    fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int INCX,  float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] += ALPHA * X[i*INCX];
}

void axpy_gpu(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    axpy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] *= X[i*INCX];
}

void mul_gpu(int N, float *X, int INCX, float *Y, int INCY) {
    mul_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void copy_kernel(int N, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = X[i*INCX];
}

void copy_gpu(int N, float *X, int INCX, float *Y, int INCY) {
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void accumulate_kernel(int N, int K, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) {
        for (int j = 0; j < K; ++j) {
            Y[i*INCY] += X[(i*K+j)*INCX];
        }
    }
}

void accumulate_gpu(int N, int K, float *X, int INCX, float *Y, int INCY) {
    accumulate_kernel<<<cuda_gridsize(N), BLOCK>>>(N, K, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void tile_repeat_kernel(int N, int K, int M, float *X, int INCX, float *Y, int INCY) {
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= M*N) return;
    int k = index % K;
    index /= K;
    int j = index % M;
    index /= M;
    int i = index;
    Y[(i*K*M+j*K+k)*INCY] = X[(i*K+k)*INCX];
}

void tile_repeat_gpu(int N, int K, int M, float *X, int INCX, float *Y, int INCY) {
    tile_repeat_kernel<<<cuda_gridsize(N*M), BLOCK>>>(N, K, M, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = X[i*INCX] + ALPHA;
}

void add_gpu(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    add_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void scale_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = X[i*INCX] * ALPHA;
}

void scale_gpu(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    scale_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = pow(X[i*INCX], ALPHA);
}

void pow_gpu(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    pow_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void min_kernel(int N, float MIN, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = fmaxf(MIN, X[i*INCX]);
}

void min_gpu(int N, float MIN, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    min_kernel<<<cuda_gridsize(N), BLOCK>>>(N, MIN, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void max_kernel(int N, float MAX, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = fminf(MAX, X[i*INCX]);
}

void max_gpu(int N, float MAX, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    max_kernel<<<cuda_gridsize(N), BLOCK>>>(N, MAX, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void padding_kernel(float* X, int w, int h, int c, int p, float* Y) {
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int pw = w + 2 * p;
    int ph = h + 2 * p;
    if (index >= pw*ph*c) return;
    int wi = index % pw;
    index /= pw;
    int hi = index % ph;
    index /= ph;
    int ci = index;
    if (wi < p || wi >= pw - p || hi < p || hi >= ph - p) Y[ci*ph*pw+hi*pw+wi] = 0;
    else Y[ci*ph*pw+hi*pw+wi] = X[ci*h*w+(hi-p)*w+(wi-p)];
}

void padding_gpu(float* X, int w, int h, int c, int p, float* Y) {
    int pw = w + 2 * p;
    int ph = h + 2 * p;
    padding_kernel<<<cuda_gridsize(ph*pw*c), BLOCK>>>(X, w, h, c, p, Y);
    check_error(hipPeekAtLastError());
}

__global__ void unrolling_kernel(float* X, int w, int h, int c, int k, int s, float* Y) {
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int yw = (w - k) / s + 1;
    int yh = (h - k) / s + 1;
    if (index >= yw*yh*k*k*c) return;
    int kj = index % k;
    index /= k;
    int ki = index % k;
    index /= k;
    int ci = index % c;
    index /= c;
    int wi = index % yw;
    index /= yw;
    int hi = index
    Y[(hi*yw+wi)*c*k*k+ci*k*k+ki*k+kj] = x[ci*h*w+(hi*s+ki)*w+wi*s+kj];
}

void unrolling_gpu(float* X, int w, int h, int c, int k, int s, float* Y) {
    int yw = (w - k) / s + 1;
    int yh = (h - k) / s + 1;

    unrolling_kernel<<<cuda_gridsize(yh*yw*k*k*c), BLOCK>>>(X, w, h, c, k, s, Y);
    check_error(hipPeekAtLastError());
}