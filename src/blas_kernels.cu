#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include "hiprand/hiprand_kernel.h"

#include "blas.h"
#include "hip/hip_runtime.h"
#include "utils.h"

#include <assert.h>

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) X[i*INCX] = ALPHA;
}

void fill_gpu(int N, float ALPHA, float *X, int INCX) {
    fill_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX);
    check_error(hipPeekAtLastError());
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int INCX,  float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] += ALPHA * X[i*INCX];
}

void axpy_gpu(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    axpy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] *= X[i*INCX];
}

void mul_gpu(int N, float *X, int INCX, float *Y, int INCY) {
    mul_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void copy_kernel(int N,  float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = X[i*INCX];
}

void copy_gpu(int N, float *X, int INCX, float *Y, int INCY) {
    copy_kernel<<<cuda_gridsize(N), BLOCK>>>(N, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = X[i*INCX] + ALPHA;
}

void add_gpu(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    add_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void scale_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = X[i*INCX] * ALPHA;
}

void scale_gpu(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    scale_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = pow(X[i*INCX], ALPHA);
}

void pow_gpu(int N, float ALPHA, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    pow_kernel<<<cuda_gridsize(N), BLOCK>>>(N, ALPHA, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void min_kernel(int N, float MIN, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = fmaxf(MIN, X[i*INCX]);
}

void min_gpu(int N, float MIN, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    min_kernel<<<cuda_gridsize(N), BLOCK>>>(N, MIN, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void max_kernel(int N, float MAX, float *X, int INCX, float *Y, int INCY) {
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < N) Y[i*INCY] = fminf(MAX, X[i*INCX]);
}

void max_gpu(int N, float MAX, float *X, int INCX, float *Y, int INCY) {
    if (Y == 0) {
        Y = X;
        INCY = INCX;
    }
    max_kernel<<<cuda_gridsize(N), BLOCK>>>(N, MAX, X, INCX, Y, INCY);
    check_error(hipPeekAtLastError());
}

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size) {
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
}

void add_bias_gpu(float *output, float *biases, int batch, int n, int size) {
    int num = n*size*batch;
    add_bias_kernel<<<cuda_gridsize(num), BLOCK>>>(output, biases, batch, n, size);
    check_error(hipPeekAtLastError());
}